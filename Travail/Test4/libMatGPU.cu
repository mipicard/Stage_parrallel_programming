#include "hip/hip_runtime.h"
/*
	MULTIPLICATION DE 2 MATRICE : M*N -> P
	Sur GPU
*/
// Librairies non GPU
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include "libMat.h" //Librairie des matrices sur CPU
#include "libMatGPU.h"


float* iniSquareGPU(const float *M,const int taille){
	float *Mg=NULL;
	if((hipMalloc((void **)&Mg,taille)) != hipSuccess){exit(EXIT_FAILURE);}
	if((hipMemcpy(Mg,M,taille,hipMemcpyHostToDevice)) != hipSuccess){exit(EXIT_FAILURE);}
	return Mg;
}


// 1ere version du multiplicateur de matrice, limité à des matrices 32*32 maximum
void multGPU1_Square(const float *M,const float *N,float *P,const int Width){
	if(Width*Width>1024){printf("\nL'algorithme actuel ne permet pas des calculs de matrices d'aires supérieur à 2^10\n");exit(EXIT_SUCCESS);}
	int taille = Width * Width * sizeof(float);
	
	//initialisation des matrices sur le GPU
	float *Mg=iniSquareGPU(M,taille),*Ng=iniSquareGPU(N,taille);
	float *Pg=NULL;
	if((hipMalloc((void **)&Pg,taille))!=hipSuccess){exit(EXIT_FAILURE);}
	
	//appel la fonction de calcul
	dim3 dimBlock(Width,Width,1), dimGrid(1,1,1); // définition de la structure 3D des blocs et de la grille
	/*
		block : structure des threads par blocs, ici en une matrice carrée, donc Width x Width x 1
		grille : ici, on n'utilise que 1 bloc de calcul, donc 1 x 1 x 1
	*/
	multGPU1_Square_aux<<<dimGrid,dimBlock>>>(Mg,Ng,Pg,Width); //La grille puis les blocs
	
	//copie de la matrice obtenue
	if((hipMemcpy(P,Pg,taille,hipMemcpyDeviceToHost)) != hipSuccess){exit(EXIT_FAILURE);}
	//libération des matrices sur le GPU
	hipFree(Mg);
	hipFree(Ng);
	hipFree(Pg);
}

__global__ void multGPU1_Square_aux(float *Mg,float *Ng,float *Pg,int Width){
	// ID des threads (ici en 2d puisque que l'on est en matrice 2d)
	int tx = threadIdx.x,ty = threadIdx.y,k;
	
	//variable somme
	float sum = 0,eMg,eNg;
	
	//calcul de chaque case de P, référencé par tx et ty
	for(k=0;k<Width;k++){
		eMg = Mg[ty*Width+k];
		eNg = Ng[k*Width+tx];
		sum+= eMg * eNg;
	}
	// inscris la valeur dans la case correspondante
	Pg[ty*Width+tx] = sum;
}


//2eme version, avec des matrices allant jusqu'à des taille de 65535*65535 de blocs de 32*32
void multGPU2_Square(const float *M,const float *N,float *P,const int Width){
	int taille = Width * Width * sizeof(float);
	
	//initialisation des matrices sur le GPU
	float *Mg=iniSquareGPU(M,taille),*Ng=iniSquareGPU(N,taille);
	float *Pg=NULL;
	if((hipMalloc((void **)&Pg,taille))!=hipSuccess){exit(EXIT_FAILURE);}
	
	//appel la fonction de calcul
		//initialisation des dimensions des blocks et de la grille
		int div = divMaxDim(Width);
		int divG = Width/div;
		if(divG>65535){printf("Erreur : la séparation en block n'est pas assez efficace pour cette dimension");exit(0);}
		dim3 dimBlock(div,div,1),dimGrid(divG,divG,1);
		
		//appel du kernel
		multGPU2_Square_aux<<<dimGrid,dimBlock>>>(Mg,Ng,Pg,Width,div);
	
	//copie de la matrice obtenue
	if((hipMemcpy(P,Pg,taille,hipMemcpyDeviceToHost)) != hipSuccess){exit(EXIT_FAILURE);}
	//libération des matrices sur le GPU
	hipFree(Mg);
	hipFree(Ng);
	hipFree(Pg);
}

__global__ void multGPU2_Square_aux(float *Mg,float *Ng,float *Pg,int Width,int nbThreadPerBlock){
	int ligne= blockIdx.y*nbThreadPerBlock + threadIdx.y, colonne= blockIdx.x*nbThreadPerBlock + threadIdx.x,k;
	float sum = 0;
	
	for(k=0;k<Width;k++){
		sum+= Mg[ligne*Width+k] * Ng[k*Width+colonne];;
	}
	
	Pg[ligne*Width+colonne] = sum;
}

int divMaxDim(const int dim){
	int answer = 32;
	while(dim%answer){answer--;}
	return answer;
}

