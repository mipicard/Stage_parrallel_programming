#include "hip/hip_runtime.h"
/*
	MULTIPLICATION DE 2 MATRICE : M*N -> P
	Sur GPU
*/
// Librairies non GPU
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include "libMat.h" //Librairie des matrices sur CPU
#include "libMatGPU.h"


float* iniSquareGPU(const float *M,const int taille){
	float *Mg=NULL;
	if((hipMalloc((void **)&Mg,taille)) != hipSuccess){exit(EXIT_FAILURE);}
	if((hipMemcpy(Mg,M,taille,hipMemcpyHostToDevice)) != hipSuccess){exit(EXIT_FAILURE);}
	return Mg;
}


// 1ere version du multiplicateur de matrice, limité à des matrices 32*32 maximum
void multGPU1_Square(const float *M,const float *N,float *P,const int Width){
	if(Width*Width>1024){printf("\nL'algorithme actuel ne permet pas des calculs de matrices d'aires supérieur à 2^10\n");exit(EXIT_SUCCESS);}
	int taille = Width * Width * sizeof(float);
	
	//initialisation des matrices sur le GPU
	float *Mg=iniSquareGPU(M,taille),*Ng=iniSquareGPU(N,taille);
	float *Pg=NULL;
	if((hipMalloc((void **)&Pg,taille))!=hipSuccess){exit(EXIT_FAILURE);}
	
	//appel la fonction de calcul
	dim3 dimBlock(Width,Width,1), dimGrid(1,1,1); // définition de la structure 3D des blocs et de la grille
	/*
		block : structure des threads par blocs, ici en une matrice carrée, donc Width x Width x 1
		grille : ici, on n'utilise que 1 bloc de calcul, donc 1 x 1 x 1
	*/
	multGPU1_Square_aux<<<dimGrid,dimBlock>>>(Mg,Ng,Pg,Width); //La grille puis les blocs
	
	//copie de la matrice obtenue
	if((hipMemcpy(P,Pg,taille,hipMemcpyDeviceToHost)) != hipSuccess){exit(EXIT_FAILURE);}
	//libération des matrices sur le GPU
	hipFree(Mg);
	hipFree(Ng);
	hipFree(Pg);
}

__global__ void multGPU1_Square_aux(float *Mg,float *Ng,float *Pg,int Width){
	// ID des threads (ici en 2d puisque que l'on est en matrice 2d)
	int tx = threadIdx.x,ty = threadIdx.y,k;
	
	//variable somme
	float sum = 0,eMg,eNg;
	
	//calcul de chaque case de P, référencé par tx et ty
	for(k=0;k<Width;k++){
		eMg = Mg[ty*Width+k];
		eNg = Ng[k*Width+tx];
		sum+= eMg * eNg;
	}
	// inscris la valeur dans la case correspondante
	Pg[ty*Width+tx] = sum;
}


//2eme version, avec des matrices allant jusqu'à des taille de 65535*65535 de blocs de 32*32
void multGPU2_Square(const float *M,const float *N,float *P,const int Width){
	int taille = Width * Width * sizeof(float);
	
	//initialisation des matrices sur le GPU
	float *Mg=iniSquareGPU(M,taille),*Ng=iniSquareGPU(N,taille);
	float *Pg=NULL;
	if((hipMalloc((void **)&Pg,taille))!=hipSuccess){exit(EXIT_FAILURE);}
	
	//appel la fonction de calcul
		//initialisation des dimensions des blocks et de la grille
		int div = divMaxDim(Width);
		int divG = Width/div;
		if(divG>65535){printf("Erreur : la séparation en block n'est pas assez efficace pour cette dimension");exit(0);}
		dim3 dimBlock(div,div,1),dimGrid(divG,divG,1);
		
		//appel du kernel
		multGPU2_Square_aux<<<dimGrid,dimBlock>>>(Mg,Ng,Pg,Width,div);
	
	//copie de la matrice obtenue
	if((hipMemcpy(P,Pg,taille,hipMemcpyDeviceToHost)) != hipSuccess){exit(EXIT_FAILURE);}
	//libération des matrices sur le GPU
	hipFree(Mg);
	hipFree(Ng);
	hipFree(Pg);
}

__global__ void multGPU2_Square_aux(float *Mg,float *Ng,float *Pg,int Width,int nbThreadPerBlock){
	int ligne= blockIdx.y*nbThreadPerBlock + threadIdx.y, colonne= blockIdx.x*nbThreadPerBlock + threadIdx.x,k;
	float sum = 0;
	
	for(k=0;k<Width;k++){
		sum+= Mg[ligne*Width+k] * Ng[k*Width+colonne];;
	}
	
	Pg[ligne*Width+colonne] = sum;
}

int divMaxDim(const int dim){
	int answer = 32;
	while(dim%answer){answer--;}
	return answer;
}

//3eme version, avec partage des donnees dans les shaders (partagé entre chaque thread d'un block) donc temps d'accès de 1/Width (~, il y a Width/div phases de calcul)
void multGPU3_Square(const float *M,const float *N,float *P,const int Width){
	int taille = Width * Width * sizeof(float);
	
	//initialisation des matrices sur le GPU
	float *Mg=iniSquareGPU(M,taille),*Ng=iniSquareGPU(N,taille);
	float *Pg=NULL;
	if((hipMalloc((void **)&Pg,taille))!=hipSuccess){exit(EXIT_FAILURE);}
	
	//appel la fonction de calcul
		//initialisation des dimensions des blocks et de la grille
		int div = divMaxDim(Width);
		int divG = Width/div;
		if(divG>65535){printf("Erreur : la séparation en block n'est pas assez efficace pour cette dimension");exit(0);}
		dim3 dimBlock(div,div,1),dimGrid(divG,divG,1);
		
		//appel du kernel
		multGPU3_Square_aux<<<dimGrid,dimBlock>>>(Mg,Ng,Pg,Width,div);
	
	//copie de la matrice obtenue
	if((hipMemcpy(P,Pg,taille,hipMemcpyDeviceToHost)) != hipSuccess){exit(EXIT_FAILURE);}
	//libération des matrices sur le GPU
	hipFree(Mg);
	hipFree(Ng);
	hipFree(Pg);
}

__global__ void multGPU3_Square_aux(float *Mg,float *Ng,float *Pg,int Width,int nbThreadPerBlock){
	float sum = 0;
	__shared__ float Mgshader[32][32]; //On met le nbThreadPerBlock maximal possible, sinon il est impossible d'utiliser cette méthode...
	__shared__ float Ngshader[32][32];
	
	int s,k,bx=blockIdx.x,by=blockIdx.x,tx=threadIdx.x,ty=threadIdx.y;
	int ligne = by*nbThreadPerBlock+ty, colonne = bx*nbThreadPerBlock+tx;
	
	for(s=0;s<(Width/nbThreadPerBlock);s++)
	{
		Mgshader[ty][tx]=Mg[ligne*Width+(s*nbThreadPerBlock + tx)];
		Ngshader[ty][tx]=Ng[colonne+Width*(s*nbThreadPerBlock + ty)];
		__syncthreads();
		
		for(k=0;k<nbThreadPerBlock;k++){
			sum+= Mgshader[ty][k] * Ngshader[k][tx];
		}
		__syncthreads();
	}
	Pg[ligne*Width+colonne] = sum;
}

