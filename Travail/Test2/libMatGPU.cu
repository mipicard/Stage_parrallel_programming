#include "hip/hip_runtime.h"
/*
	MULTIPLICATION DE 2 MATRICE : M*N -> P
	Sur GPU
*/
// Librairies non GPU
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include "libMat.h" //Librairie des matrices sur CPU
#include "libMatGPU.h"

void multGPU1_Square(const float *M,const float *N,float *P,const int Width){
	if(Width>512){printf("L'algorithme actuel ne permet pas des calculs de matrices de taille supérieur à 512");exit(EXIT_SUCCESS);}
	int taille = Width * Width * sizeof(float);
	
	//initialisation des matrices sur le GPU
	float *Mg=iniSquareGPU(M,taille),*Ng=iniSquareGPU(N,taille);
	float *Pg=NULL;
	if((hipMalloc((void **)&Pg,taille))!=hipSuccess){exit(EXIT_FAILURE);}
	
	//appel la fonction de calcul
	dim3 dimBlock(Width,Width,1), dimGrid(1,1,1); // définition de la structure 3D des blocs et de la grille
	/*
		block : structure des threads par blocs, ici en une matrice carrée, donc Width x Width x 1
		grille : ici, on n'utilise que 1 bloc de calcul, donc 1 x 1 x 1
	*/
	multGPU1_Square_aux<<<dimBlock,dimGrid>>>(Mg,Ng,Pg,Width);
	//copie de la matrice obtenue
	if((hipMemcpy(P,Pg,taille,hipMemcpyDeviceToHost)) != hipSuccess){exit(EXIT_FAILURE);}
	//libération des matrices sur le GPU
	hipFree(Mg);
	hipFree(Ng);
	hipFree(Pg);
}

float* iniSquareGPU(const float *M,const int taille){
	float *Mg=NULL;
	if((hipMalloc((void **)&Mg,taille)) != hipSuccess){exit(EXIT_FAILURE);}
	if((hipMemcpy(Mg,M,taille,hipMemcpyHostToDevice)) != hipSuccess){exit(EXIT_FAILURE);}
	return Mg;
}

__global__ void multGPU1_Square_aux(const float *Mg,const float *Ng,float *Pg,const int Width){
	// ID des threads (ici en 2d puisque que l'on est en matrice 2d)
	int tx = threadIdx.x,ty = threadIdx.y,k;
	
	//variable somme
	float sum = 0,eMg,eNg;
	
	//calcul de chaque case de P, référencé par tx et ty
	for(k=0;k<Width;k++){
		eMg = Mg[ty*Width+k];
		eNg = Ng[k*Width+tx];
		sum+= eMg * eNg;
	}
	// inscris la valeur dans la case correspondante
	Pg[ty*Width+tx] = sum;
}
