#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include "DEBUG.h"

#include "Element.h"
#include "MatriceGPU.h"

__host__ __device__ inline static unsigned int positionElement(const unsigned int i,const unsigned int j,const MatriceGPU *m){return i*m->dimension+j;}

inline static void pointeurNonAlloue(const void *pointeur){if(!(pointeur == NULL)){exit(84);}}

inline static void cudaFail(hipError_t e){if(e != hipSuccess){exit(85);}}

inline void matriceNonInitialiseGPU(const MatriceGPU *pointeur){pointeurNonAlloue(pointeur);pointeurNonAlloue(pointeur->matrice);}

static int divMaxDim(const int dim){
	int answer = 32;
	while(dim%answer){answer--;}
	return answer;
}

MatriceGPU *initialiserMatriceGPU(const unsigned long taille){
	const unsigned long qtMemory = taille*taille*sizeof(Element);
	MatriceGPU *m = NULL;
	cudaFail(hipMalloc((void **)&m,sizeof(MatriceGPU)));
	m->matrice = NULL;
	cudaFail(hipMalloc((void **)&(m->matrice),qtMemory));
	cudaFail(hipMemset(m->matrice,ZERO_ELEMENT,qtMemory));
	return m;
}

void freeMatriceGPU(MatriceGPU *m){
	if(m!=NULL){
		hipFree(m->matrice);
		hipFree(m);
		m=NULL;
	}
}

__global__ static void additionMatriceGPU_Kernel(const MatriceGPU *m1,const MatriceGPU *m2,MatriceGPU *resultat,const int nbThreadPerBlock){
	unsigned long ligne = blockIdx.y*nbThreadPerBlock+threadIdx.y,colonne = blockIdx.x*nbThreadPerBlock +threadIdx.x;
	resultat->matrice[positionElement(ligne,colonne,resultat)]=additionElement(m1->matrice[positionElement(ligne,colonne,m1)],m2->matrice[positionElement(ligne,colonne,m2)]);
}

void additionMatriceGPU(const MatriceGPU *m1,const MatriceGPU *m2,MatriceGPU *resultat){
	const unsigned long dim=resultat->dimension;
	int div = divMaxDim(dim);
	int divG = dim/div;
	dim3 dimBlock(div,div,1),dimGrid(divG,divG,1);
	
	additionMatriceGPU_Kernel<<<dimGrid,dimBlock>>>(m1,m2,resultat,div);
	
}

__global__ static void multiplicationMatriceGPU_Kernel(const MatriceGPU *m1,const MatriceGPU *m2,MatriceGPU *resultat,const int nbThreadPerBlock){
	unsigned long ligne= blockIdx.y*nbThreadPerBlock + threadIdx.y, colonne= blockIdx.x*nbThreadPerBlock + threadIdx.x;
	Element sum = ZERO_ELEMENT;
	
	for(int k=0;k<resultat->dimension;k++){
		sum = additionElement(sum,multiplicationElement(m1->matrice[ligne*m1->dimension+k],m2->matrice[k*m2->dimension+colonne]));;
	}
	
	resultat->matrice[positionElement(ligne,colonne,resultat)] = sum;
}

void multiplicationMatriceGPU(const MatriceGPU *m1,const MatriceGPU *m2,MatriceGPU *resultat){
	const unsigned long dim=resultat->dimension;
	int div = divMaxDim(dim);
	int divG = dim/div;
	dim3 dimBlock(div,div,1),dimGrid(divG,divG,1);
	
	multiplicationMatriceGPU_Kernel<<<dimGrid,dimBlock>>>(m1,m2,resultat,div);
}

int matriceEqualGPU(const MatriceGPU *m1,const MatriceGPU *m2){
	int res=1,i=0,j;
	while(res && i<m1->dimension){
		j=0;
		while(res && j<m1->dimension){
			res = equalElement(m1->matrice[positionElement(i,j,m1)],m2->matrice[positionElement(i,j,m2)]);
			j++;
		}
		i++;
	}
	return res;
}

void fillRandomMatriceGPU(MatriceGPU *m){
	for(int i=0;i<m->dimension;i++){
		for(int j=0;j<m->dimension;j++)
			m->matrice[positionElement(i,j,m)] = randomElement();		
	}
}
