#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include "DEBUG.h"

#include "Element.h"
#include "MatriceGPU.h"


//__device__ float equal;

__host__ __device__ inline static unsigned int positionElement(const unsigned int i,const unsigned int j,const MatriceGPU *m){return i*m->dimension+j;}

inline static void pointeurNonAlloue(const void *pointeur){if(pointeur == NULL){exit(84);}}

inline static void cudaFail(hipError_t e){if(e != hipSuccess){exit(85);}}

inline void matriceNonInitialiseGPU(const MatriceGPU *pointeur){pointeurNonAlloue(pointeur);pointeurNonAlloue(pointeur->matrice);}

static int divMaxDim(const int dim){
	int answer = 32;
	while(dim%answer){answer--;}
	return answer;
}

MatriceGPU *initialiserMatriceGPU(const unsigned long taille){
	const unsigned long qtMemory = taille*taille*sizeof(Element);
	MatriceGPU *m = (MatriceGPU *)malloc(sizeof(MatriceGPU));
	pointeurNonAlloue(m);
	cudaFail(hipMalloc((void **)&m->matrice,qtMemory));
	m->dimension=taille;
	return m;
}

void freeMatriceGPU(MatriceGPU *m){
	if(m!=NULL){
		hipFree(m->matrice);
		free(m);
		m=NULL;
	}
}

__global__ static void additionMatriceGPU_Kernel(const MatriceGPU m1,const MatriceGPU m2,MatriceGPU resultat,const int nbThreadPerBlock){
	unsigned long ligne = blockIdx.y*nbThreadPerBlock+threadIdx.y,colonne = blockIdx.x*nbThreadPerBlock +threadIdx.x;
	resultat.matrice[positionElement(ligne,colonne,&resultat)]=additionElement(m1.matrice[positionElement(ligne,colonne,&m1)],m2.matrice[positionElement(ligne,colonne,&m2)]);
}

void additionMatriceGPU(const MatriceGPU *m1,const MatriceGPU *m2,MatriceGPU *resultat){
	const unsigned long dim=resultat->dimension;
	int div = divMaxDim(dim);
	int divG = dim/div;
	dim3 dimBlock(div,div,1),dimGrid(divG,divG,1);
	
	additionMatriceGPU_Kernel<<<dimGrid,dimBlock>>>(*m1,*m2,*resultat,div);
	
}

#ifdef GPU_OPTI
__global__ static void multiplicationMatriceGPU_Kernel(const MatriceGPU m1,const MatriceGPU m2,MatriceGPU resultat,const int nbThreadPerBlock){
	Element sum = ZERO_ELEMENT;
	__shared__ Element Mgshader[32][32]; //On met le nbThreadPerBlock maximal possible, sinon il est impossible d'utiliser cette méthode...
	__shared__ Element Ngshader[32][32];
	
	int bx=blockIdx.x,by=blockIdx.x,tx=threadIdx.x,ty=threadIdx.y;
	int ligne = by*nbThreadPerBlock+ty, colonne = bx*nbThreadPerBlock+tx;
	unsigned long Width = resultat.dimension;
	
	for(int s=0;s<(Width/nbThreadPerBlock);s++)
	{
		Mgshader[ty][tx]=m1.matrice[ligne*Width+(s*nbThreadPerBlock + tx)];
		Ngshader[ty][tx]=m1.matrice[colonne+Width*(s*nbThreadPerBlock + ty)];
		__syncthreads();
		
		for(int k=0;k<nbThreadPerBlock;k++){
			sum=additionElement(sum,multiplicationElement(Mgshader[ty][k],Ngshader[k][tx]));
		}
		__syncthreads();
	}
	resultat.matrice[ligne*Width+colonne] = sum;
}
#else
__global__ static void multiplicationMatriceGPU_Kernel(const MatriceGPU m1,const MatriceGPU m2,MatriceGPU resultat,const int nbThreadPerBlock){
	unsigned long ligne= blockIdx.y*nbThreadPerBlock + threadIdx.y, colonne= blockIdx.x*nbThreadPerBlock + threadIdx.x;
	Element sum = ZERO_ELEMENT;
	
	for(int k=0;k<resultat.dimension;k++){
		sum = additionElement(sum,multiplicationElement(m1.matrice[ligne*m1.dimension+k],m2.matrice[k*m2.dimension+colonne]));;
	}
	
	resultat.matrice[positionElement(ligne,colonne,&resultat)] = sum;
}
#endif

void multiplicationMatriceGPU(const MatriceGPU *m1,const MatriceGPU *m2,MatriceGPU *resultat){
	const unsigned long dim=resultat->dimension;
	int div = divMaxDim(dim);
	int divG = dim/div;
	dim3 dimBlock(div,div,1),dimGrid(divG,divG,1);
	
	multiplicationMatriceGPU_Kernel<<<dimGrid,dimBlock>>>(*m1,*m2,*resultat,div);
	hipDeviceSynchronize();
}
/*
__global__ static void matriceEqualGPU_Kernel(const MatriceGPU m1,const MatriceGPU m2,const int nbThreadPerBlock){
	__shared__ float answer;
	answer = 0.0;
	unsigned long ligne= blockIdx.y*nbThreadPerBlock + threadIdx.y, colonne= blockIdx.x*nbThreadPerBlock + threadIdx.x;
	answer += (!(equalElement(m1.matrice[positionElement(ligne,colonne,&m1)],m2.matrice[positionElement(ligne,colonne,&m2)])));
	__syncthreads();
	equal+=answer;
	
}

int matriceEqualGPU(const MatriceGPU *m1,const MatriceGPU *m2){
	equal = 0.0;
	const unsigned long dim=m1->dimension;
	int div = divMaxDim(dim);
	int divG = dim/div;
	dim3 dimBlock(div,div,1),dimGrid(divG,divG,1);
	
	matriceEqualGPU_Kernel<<<dimGrid,dimBlock>>>(*m1,*m2,div);
	return (equal==0.0);
}
*/
void fillRandomMatriceGPU(MatriceGPU *m){
	unsigned long qtMemory = m->dimension*m->dimension*sizeof(Element);
	Element *sub = (Element *) malloc(qtMemory);
	pointeurNonAlloue(sub);
	for(int i=0;i<m->dimension;i++){
		for(int j=0;j<m->dimension;j++)
			sub[positionElement(i,j,m)]=randomElement();		
	}
	hipMemcpy(m->matrice,sub,qtMemory,hipMemcpyHostToDevice);
}
